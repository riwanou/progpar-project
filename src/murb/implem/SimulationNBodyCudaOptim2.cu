#include "hip/hip_runtime.h"
#include <string>

#include "SimulationNBodyCudaOptim2.hpp"

#define CUDA_CHECK(call)                                                                                               \
    {                                                                                                                  \
        hipError_t err = call;                                                                                        \
        if (err != hipSuccess) {                                                                                      \
            fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);                 \
            exit(err);                                                                                                 \
        }                                                                                                              \
    }

__global__ void kernel_cuda_optim2(cudaPackedAoS_t<float> *inBodies, accAoS_t<float> *outAccelerations,
                                   const unsigned int nbBodies, const float soft, const float G, const int offset)
{
    const int sizePass = 1536;
    const int nbPass = (nbBodies + sizePass - 1) / sizePass;

    static __shared__ cudaPackedAoS_t<float> shBodies[sizePass];

    const int iBody = (blockDim.x * blockIdx.x + threadIdx.x) * 2;
    const int iBody1 = (blockDim.x * blockIdx.x + threadIdx.x) * 2 + 1;
    const float softSquared = soft * soft;

    float4 a = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float4 a1 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    float4 q = make_float4(inBodies[iBody].qx, inBodies[iBody].qy, inBodies[iBody].qz, 0.0f);
    float4 q1 = make_float4(inBodies[iBody1].qx, inBodies[iBody1].qy, inBodies[iBody1].qz, 0.0f);

    // shared memory is too small to contains all the bodies
    // acumulate the acceleration in multiple passes
    for (int pass = 0; pass < nbPass; pass++) {
        const int startIdx = pass * sizePass;
        const int endIdx = min((pass + 1) * sizePass, nbBodies);
        unsigned int shIdx = 0;

        // load in shared memory
        shBodies[threadIdx.x] = inBodies[startIdx + threadIdx.x];
        shBodies[threadIdx.x + 768] = inBodies[startIdx + threadIdx.x + 768];
        __syncthreads();

        for (int jBody = startIdx; jBody < endIdx; jBody++) {
            float4 shBody = make_float4(shBodies[shIdx].qx, shBodies[shIdx].qy, shBodies[shIdx].qz, shBodies[shIdx].m);

            float rijx = shBody.x - q.x;
            float rijy = shBody.y - q.y;
            float rijz = shBody.z - q.z;

            float rijSquared = rijx * rijx + rijy * rijy + rijz * rijz + softSquared;
            float revSqrt = rsqrtf(rijSquared);
            float rsqrt3 = revSqrt * revSqrt * revSqrt;
            float ai = G * shBody.w * rsqrt3;

            a.x += ai * rijx;
            a.y += ai * rijy;
            a.z += ai * rijz;

            rijx = shBody.x - q1.x;
            rijy = shBody.y - q1.y;
            rijz = shBody.z - q1.z;

            rijSquared = rijx * rijx + rijy * rijy + rijz * rijz + softSquared;
            revSqrt = rsqrtf(rijSquared);
            rsqrt3 = revSqrt * revSqrt * revSqrt;
            ai = G * shBody.w * rsqrt3;

            a1.x += ai * rijx;
            a1.y += ai * rijy;
            a1.z += ai * rijz;

            shIdx++;
        }

        __syncthreads();
    }

    outAccelerations[iBody].ax = a.x;
    outAccelerations[iBody].ay = a.y;
    outAccelerations[iBody].az = a.z;
    outAccelerations[iBody1].ax = a1.x;
    outAccelerations[iBody1].ay = a1.y;
    outAccelerations[iBody1].az = a1.z;
}

SimulationNBodyCudaOptim2::SimulationNBodyCudaOptim2(const unsigned long nBodies, const std::string &scheme,
                                                     const float soft, const unsigned long randInit)
    : SimulationNBodyInterface(nBodies, scheme, soft, randInit)
{
    this->flopsPerIte = (20.f * (float)this->getBodies().getN() * (float)this->getBodies().getN()) +
                        (9.0f * (float)this->getBodies().getN());
    this->accelerations.resize(this->getBodies().getN());
    this->packedBodies.resize(this->getBodies().getN());

    CUDA_CHECK(hipMalloc(&cudaAccelerations, this->getBodies().getN() * sizeof(accAoS_t<float>)));
    CUDA_CHECK(hipMalloc(&cudaBodies, this->getBodies().getN() * sizeof(cudaPackedAoS_t<float>)));
}

void SimulationNBodyCudaOptim2::initIteration()
{
    std::vector<dataAoS_t<float>> bodies = this->getBodies().getDataAoS();

    for (unsigned long iBody = 0; iBody < this->getBodies().getN(); iBody++) {
        dataAoS_t<float> body = bodies[iBody];
        this->packedBodies[iBody].qx = body.qx;
        this->packedBodies[iBody].qy = body.qy;
        this->packedBodies[iBody].qz = body.qz;
        this->packedBodies[iBody].m = body.m;
    }
}

void SimulationNBodyCudaOptim2::computeBodiesAcceleration()
{
    dim3 block(768);
    int nbBlocks = (this->getBodies().getN() + block.x - 1) / block.x;
    nbBlocks = (nbBlocks + 1) / 2;
    dim3 grid(nbBlocks);

    CUDA_CHECK(hipMemcpy(this->cudaBodies, this->packedBodies.data(),
                          this->getBodies().getN() * sizeof(cudaPackedAoS_t<float>), hipMemcpyHostToDevice));

    kernel_cuda_optim2<<<grid, block>>>(this->cudaBodies, this->cudaAccelerations, this->getBodies().getN(), this->soft,
                                        this->G, nbBlocks * block.x);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(this->accelerations.data(), this->cudaAccelerations,
                          this->getBodies().getN() * sizeof(accAoS_t<float>), hipMemcpyDeviceToHost));
}

void SimulationNBodyCudaOptim2::computeOneIteration()
{
    this->initIteration();
    this->computeBodiesAcceleration();
    // time integration
    this->bodies.updatePositionsAndVelocities(this->accelerations, this->dt);
}
